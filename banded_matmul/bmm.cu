#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>

__global__ void bandedMatMul(const Matrix &t0, const BandedMatrix &t1,
                             Matrix &t2) {

  float sum = t0.data[0];
}

void run(int nBand) {
  const int n0 = 1024;
  const int n1 = 1024;
  const int n2 = nBand;

  Matrix T0(n0, n1);
  BandedMatrix T1(n1, n2);
  Matrix T2(T1.width(), n1);

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  T0.init(3);
  T1.init(4);
  T2.init(0);

  // Launch the kernel
  dim3 threads(16, 16, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);

  bandedMatMul<<<blocks, threads>>>(T0, T1, T2);

  CHECK(hipDeviceSynchronize());

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  CHECK(hipGetDevice(&deviceId));
  run(16);
  return 0;
}