#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


hipError_t CHECK(hipError_t res) {
  if (hipSuccess != res) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(res));
    exit(EXIT_FAILURE);
  }
  return res;
}


int main(int argc, const char **argv) {
  int deviceId;
  CHECK(hipGetDevice(&deviceId));

  return 0;
}