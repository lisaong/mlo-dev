#include "utils.h"
#include <hip/hip_runtime.h>

void run(int nBand) {
  const int n0 = 1024;
  const int n1 = 1024;
  const int n2 = nBand;

  // dense size
  float *T0, *T1, *T2;
  const int n2Full = n0 + n2;

  CHECK(hipMallocManaged(&T0, n0 * n1 * sizeof(float)));
  CHECK(hipMallocManaged(&T1, n0 * n2Full * sizeof(float)));
  CHECK(hipMallocManaged(&T2, n2Full * sizeof(float)));

  hipFree(T0);
  hipFree(T1);
  hipFree(T2);
}

int main(int argc, const char **argv) {
  int deviceId;
  CHECK(hipGetDevice(&deviceId));
  run(16);
  return 0;
}