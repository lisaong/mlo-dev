#include "hip/hip_runtime.h"
// Naive banded matrix multiplication
#include <cstdint>
#include <hip/hip_runtime.h>

// #define DEBUG 1
#include "constants.h"
#include "utils.h"

__global__ void bandedMatMul_Naive(int n0, int n1, int n2, float *t0,
                                   const float *t1, const float *t2) {

  int i, j, k;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.y * gridDim.y) {
      for (k = 0; k < n2 && (i + k) < n0; ++k) {
        t0[i * n1 + j] += t1[i * n2 + k] * t2[(i + k) * n1 + j];
      }
    }
  }
}

void run(int deviceId) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);             // output
  BandedMatrix<float> T1(n0, kBandDim); // input
  Matrix<float> T2(T1.columns(), n1);   // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);
  initWith<<<blocks, threads>>>(11.0f, T0.data, T0.rows(), T0.columns());
  initBandedWith<<<blocks, threads>>>(22.0f, T1.data, T1.rows(), T1.columns(),
                                      T1.band());
  initWith<<<blocks, threads>>>(33.0f, T2.data, T2.rows(), T2.columns());
  CHECK(hipDeviceSynchronize());

  // Verify
  bandedMatMul_Naive<<<blocks, threads>>>(n0, n1, n2, T0.data, T1.data,
                                          T2.data);
  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    // Try different block sizes
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#thread-and-block-heuristics
    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = kMaxBlockDim / blockDim;
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y);

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {
          bandedMatMul_Naive<<<blocks, threads>>>(n0, n1, n2, T0.data, T1.data,
                                                  T2.data);

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  run(deviceId);
  return 0;
}