#include "utils.h"
#include <hip/hip_runtime.h>

void run(int nBand) {
  const int n0 = 1024;
  const int n1 = 1024;
  const int n2 = nBand;

  Matrix T0(n0, n1);
  BandedMatrix T1(n1, n2);
  Matrix T2(T1.width(), n1);

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  T0.init(3);
  T1.init(4);
  T2.init(0);

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  CHECK(hipGetDevice(&deviceId));
  run(16);
  return 0;
}