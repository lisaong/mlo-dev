#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>

void bandedMatMul_CPU(int n0, int n1, int n2, float *t0, const float *t1, const float *t2) {
  /*
    for i in range(n0):
        for j in range(n1):
            for k in range(n2):
                t0[i, j] += t1[i, k] * t2[i + k, j]
  */
  int i, j, k;
  for (i = 0; i < n0; ++i) {
    for (j = 0; j < n1; ++j) {
      for (k = 0; k < n2; ++k) {
        t0[i * n1 + j] += t1[i * n2 + k] * t2[(i + k) * n1 + j];
      }
    }
  }
}

__global__ void bandedMatMul_Naive(int n0, int n1, int n2, float *t0, const float *t1,
                                   const float *t2) {

  int i, j, k;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.x * gridDim.x) {
      for (k = blockIdx.z * blockDim.z + threadIdx.z; k < n2;
           k += blockDim.z * gridDim.z) {
        t0[i * n1 + j] += t1[i * n2 + k] * t2[(i + k) * n1 + j];
      }
    }
  }
}

void run(int nBand) {
  const int n0 = 1024;
  const int n1 = 1024;
  const int n2 = nBand;

  Matrix T0(n0, n1);           // output
  BandedMatrix T1(n1, n2);     // input
  Matrix T2(T1.columns(), n1); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  T0.init(3);
  T1.init(4);
  T2.init(0);

  dim3 threads(16, 16, 16);
  dim3 blocks(n0 / threads.x, n1 / threads.y, n2 / threads.z);

  bandedMatMul_Naive<<<blocks, threads>>>(n0, n1, n2, T0.data, T1.data,
                                          T2.data);

  CHECK(hipDeviceSynchronize());

  Matrix T0_CPU(n0, n1);
  T0_CPU.data = reinterpret_cast<float*>(malloc(T0_CPU.size()));
  T0_CPU.init(3);

  bandedMatMul_CPU(n0, n1, n2, T0_CPU.data, T1.data, T2.data);

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
  free(T0_CPU.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  CHECK(hipGetDevice(&deviceId));
  run(16);
  return 0;
}