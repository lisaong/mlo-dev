#include "hip/hip_runtime.h"
// Packed B implementation of banded matrix multiplication
// where the B matrix is packed to improve locality:
//
// [ x 0 0 0 0 0 0 0 ]      [ x x x x x x x x ]
// [ x x 0 0 0 0 0 0 ]  =>  [ x x x x x x x 0 ]  B[1, 7] = 0
// [ x x x 0 0 0 0 0 ]      [ x x x x x x 0 0 ]  B[2, 6] = B[2, 7] = 0
// [ 0 x x x 0 0 0 0 ]
// [ 0 0 x x x 0 0 0 ]
// [ 0 0 0 x x x 0 0 ]
// [ 0 0 0 0 x x x 0 ]
// [ 0 0 0 0 0 x x x ]
//
// This assumes the A-matrix is banded:
//
// [ x x x 0 0 0 0 0 ]      [ x x x ]
// [ 0 x x x 0 0 0 0 ]      [ x x x ]
// [ 0 0 x x x 0 0 0 ]      [ x x x ]
// [ 0 0 0 x x x 0 0 ]  =>  [ x x x ]
// [ 0 0 0 0 x x x 0 ]      [ x x x ]
// [ 0 0 0 0 0 x x x ]      [ x x x ]
// [ 0 0 0 0 0 0 x x ]      [ x x 0 ]  A[6, 2] = 0
// [ 0 0 0 0 0 0 0 x ]      [ x 0 0 ]  A[7, 1] = A[7, 2] = 0

#include <assert.h>
#include <cstdint>

#include "utils.h"
#include <hip/hip_runtime.h>

// #define PREFETCH 1 // doesn't help
#define DEVICE_INIT 1

#define DEBUG 1
#if DEBUG
constexpr uint32_t N = 16;
#else
constexpr uint32_t N = 1024;
#endif // DEBUG

constexpr uint32_t kBandDim = N;
constexpr uint32_t kBlockDim = 16;
constexpr uint32_t kMaxBlockDim = 1024;
constexpr uint32_t kNumberOfOps = 2 * N * N * N;
constexpr uint32_t kMillisecondsInSeconds = 1000;
constexpr uint32_t kTimeLimit = 10 * kMillisecondsInSeconds;

__global__ void initWith(float num, float *a, int rows, int columns) {

  int i, j;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < rows;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < columns;
         j += blockDim.y * gridDim.y) {
      a[i * columns + j] = num;
    }
  }
}

__global__ void initBandedWith(float num, float *a, int rows, int columns,
                               int band) {

  int i, j;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < rows;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < band;
         j += blockDim.y * gridDim.y) {

      if ((i + j) < columns) {
        a[i * band + j] = num;
      } else {
        // zero out the lower right triangle
        a[i * band + j] = 0;
      }
    }
  }
}

__global__ void initTransposeBandedWith(float num, float *a, int rows,
                                        int columns, int band) {

  int i, j;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < band;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < columns;
         j += blockDim.y * gridDim.y) {

      if ((i + j) < rows) {
        a[i * columns + j] = num;
      } else {
        // zero out the lower right triangle
        a[i * columns + j] = 0;
      }
    }
  }
}

__global__ void bandedMatMul_PackedB(int n0, int n1, int n2, float *t0,
                                     const float *t1, const float *t2) {

  int i, j;
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.y * gridDim.y) {

      for (int k = 0; k < n2; ++k) {
        t0[i * n1 + j] += t1[i * n2 + k] * t2[k * n1 + j];
      }
    }
  }
}

bool checkCorrectness(int n0, int n1, int n2, const Matrix &T0,
                      const BandedMatrix &T1,
                      const TransposedBandedMatrix &T2) {
  Matrix T0_CPU(n0, n1);
  Matrix T2_CPU(T2.rows(), T2.columns());

  T0_CPU.data = reinterpret_cast<float *>(malloc(T0_CPU.size()));
  T0_CPU.init(0.0f);
  T2_CPU.data = reinterpret_cast<float *>(malloc(T2_CPU.size()));
  T2_CPU.init(33.0f);

  bandedMatMul_CPU(n0, n1, n2, T0_CPU.data, T1.data, T2_CPU.data);

#if DEBUG
  std::cout << "T0_CPU: " << std::endl;
  T0_CPU.print();
  std::cout << "T0: " << std::endl;
  T0.print();
#endif // DEBUG

  bool result = T0_CPU == T0;
  if (result) {
    std::cout << "Values match" << std::endl;
  } else {
    std::cerr << "Values do not match" << std::endl;
  }

  free(T0_CPU.data);
  free(T2_CPU.data);
  return result;
}

bool verify() {

  // n0: number of rows in T0 and T1
  // n1: number of columns in T0 and T2
  // n2: inner or shared dimension, i.e.
  //  number of columns in T1 and number of rows in T2

  const int n0 = N;
  const int n1 = N;
  const int n2 = kBandDim;

  Matrix T0(n0, n1);                               // output
  BandedMatrix T1(n0, n1, n2);                     // input
  TransposedBandedMatrix T2(T1.columns(), n1, n2); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  dim3 threads(kBlockDim, kBlockDim, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);

#if DEVICE_INIT
  initWith<<<blocks, threads>>>(0.0f, T0.data, T0.rows(), T0.columns());
  initBandedWith<<<blocks, threads>>>(22.0f, T1.data, T1.rows(), T1.columns(),
                                      T1.band());
  initTransposeBandedWith<<<blocks, threads>>>(33.0f, T2.data, T2.rows(),
                                               T2.columns(), T2.band());
  CHECK(hipDeviceSynchronize());

#if DEBUG
  std::cout << "T1: " << std::endl;
  T1.print();
  std::cout << "T2: " << std::endl;
  T2.print();
#endif

#else
  T0.init(0.0f);
  T1.init(22.0f);
  T2.init(33.0f);
#endif

  bandedMatMul_PackedB<<<blocks, threads>>>(n0, n1, n2, T0.data, T1.data,
                                            T2.data);
  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  hipMemPrefetchAsync(T0.data, T0.size(), hipCpuDeviceId);
  bool result = checkCorrectness(n0, n1, n2, T0, T1, T2);

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);

  return result;
}

void benchmark(int deviceId) {
  // Runs the function until 10 seconds has elapsed

  hipEvent_t _start;
  hipEvent_t _stop;
  hipEventCreate(&_start);
  hipEventCreate(&_stop);

  const int n0 = N;
  const int n1 = N;
  const int n2 = kBandDim;

  Matrix T0(n0, n1);                               // output
  BandedMatrix T1(n0, n1, n2);                     // input
  TransposedBandedMatrix T2(T1.columns(), n1, n2); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

#if DEVICE_INIT

  dim3 threadsInit(kBlockDim, kBlockDim, 1);
  dim3 blocksInit(n0 / threadsInit.x, n1 / threadsInit.y, 1);

  initWith<<<threadsInit, threadsInit>>>(0.0f, T0.data, T0.rows(),
                                         T0.columns());
  initBandedWith<<<threadsInit, threadsInit>>>(22.0f, T1.data, T1.rows(),
                                               T1.columns(), T1.band());
  initTransposeBandedWith<<<threadsInit, threadsInit>>>(
      33.0f, T2.data, T2.rows(), T2.columns(), T2.band());
  CHECK(hipDeviceSynchronize());
#else
  T0.init(0.0f);
  T1.init(22.0f);
  T2.init(33.0f);
#endif // DEVICE_INIT

#if PREFETCH
  hipMemPrefetchAsync(T0.data, T0.size(), deviceId);
  hipMemPrefetchAsync(T1.data, T1.size(), deviceId);
  hipMemPrefetchAsync(T2.data, T2.size(), deviceId);
#endif // PREFETCH

  for (uint32_t blockDim = kBlockDim; blockDim <= kMaxBlockDim;
       blockDim += kBlockDim) {

    dim3 threads(blockDim, blockDim, 1);
    dim3 blocks(n0 / threads.x, n1 / threads.y, 1);

    double elapsedTimeMilliseconds = 0.0f;
    uint64_t iterations = 0;
    float duration = 0.0f;

    hipEventRecord(_start);
    while (elapsedTimeMilliseconds < kTimeLimit) {
      bandedMatMul_PackedB<<<blocks, threads>>>(n0, n1, n2, T0.data, T1.data,
                                                T2.data);
      hipDeviceSynchronize();
      hipEventRecord(_stop);
      hipEventSynchronize(_stop);
      hipEventElapsedTime(&duration, _start, _stop);
      elapsedTimeMilliseconds += duration;
      iterations++;
    }

    const double flops = iterations * kNumberOfOps /
                         (elapsedTimeMilliseconds / kMillisecondsInSeconds);
    std::cout << "Blocksize: " << blockDim << ", Iterations: " << iterations
              << ", FLOPS: " << flops << ", GFLOPS: " << flops / 1e9
              << std::endl;
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);

  hipEventDestroy(_start);
  hipEventDestroy(_stop);
}

int main(int argc, const char **argv) {
  int deviceId;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (verify()) {
    benchmark(deviceId);
  }
  return 0;
}