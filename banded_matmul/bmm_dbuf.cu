#include "hip/hip_runtime.h"
// Double-buffered banded matrix multiplication
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdint>
// #include <cuda/pipeline>
#include <hip/hip_runtime.h>

// #define DEBUG 1
#include "constants.h"
#include "utils.h"

// https://developer.nvidia.com/blog/cooperative-groups/
namespace cg = cooperative_groups;

enum class Strategy { SynchronousCopy = 0, AsynchronousCopy = 1 };

__global__ void bandedMatMul_syncCopy(int n0, int n1, int n2, float *t0,
                                      const float *t1, const float *t2,
                                      int tile) {

  int i, j, k, jj;

  auto cta = cg::this_thread_block();

  // load the t0 and t1 sub-matrices into shared memory
  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size() * tile];

  const auto rowStart = blockIdx.x * blockDim.x + threadIdx.x;
  const auto rowStride = blockDim.x * gridDim.x;
  const auto colStart = blockIdx.y * blockDim.y + threadIdx.y;
  const auto colStride = blockDim.y * gridDim.y;

  for (i = rowStart; i < n0; i += rowStride) {
    for (j = colStart; j * tile < n1; j += colStride) {

      // for each thread, copy a column-tile of t0 and t1 into shared memory
      for (jj = 0; jj < tile; ++jj) {
        const auto smemIdx =
            threadIdx.x * blockDim.y * tile + threadIdx.y * tile + jj;
        t0_s[smemIdx] = t0[i * n1 + j * tile + jj];
        t1_s[smemIdx] = t1[i * n2 + j * tile + jj];
      }
    }
  }
  cta.sync();

  // compute
  for (i = rowStart; i < n0; i += rowStride) {
    for (j = colStart; j * tile < n1; j += colStride) {
      for (jj = 0; jj < tile; ++jj) {
        const auto smemIdx =
            threadIdx.x * blockDim.y * tile + threadIdx.y * tile + jj;

        // treat t2 as column major
        for (k = 0; i + k < n1; ++k) {
          t0_s[smemIdx] += t1_s[smemIdx] * t2[(i + k) + (j * tile + jj) * n2];
        }
        cta.sync();

        // write back to global memory
        t0[i * n1 + j * tile + jj] = t0_s[smemIdx];
      }
    }
  }
}

__global__ void bandedMatMul_asyncCopy(int n0, int n1, int n2, float *t0,
                                       const float *t1, const float *t2,
                                       int tile) {

  int i, j, k, jj;

  auto cta = cg::this_thread_block();

  // load the t0 and t1 sub-matrices into shared memory
  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size() * tile];

  const auto rowStart = blockIdx.x * blockDim.x + threadIdx.x;
  const auto rowStride = blockDim.x * gridDim.x;
  const auto colStart = blockIdx.y * blockDim.y + threadIdx.y;
  const auto colStride = blockDim.y * gridDim.y;

  i = rowStart;
  j = colStart;
  cg::memcpy_async(cta, t0_s, &t0[i * n1 + j * tile],
                   sizeof(float) * tile * cta.size());
  cg::memcpy_async(cta, t1_s, &t1[i * n2 + j * tile],
                   sizeof(float) * tile * cta.size());
  cg::wait(cta);

  for (i = rowStart; i < n0; i += rowStride) {
    for (j = colStart; j * tile < n1; j += colStride) {

      // compute
      for (jj = 0; jj < tile; ++jj) {
        const auto smemIdx =
            threadIdx.x * blockDim.y * tile + threadIdx.y * tile + jj;

        // treat t2 as column major
        for (k = 0; i + k < n1; ++k) {
          t0_s[smemIdx] += t1_s[smemIdx] * t2[(i + k) + (j * tile + jj) * n2];
        }

        // write back to global memory
        t0[i * n1 + j * tile + jj] = t0_s[smemIdx];
      }
    }
  }
}

void run(int deviceId, Strategy strategy) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);                                 // output
  BandedMatrix<float> T1(n0, kBandDim);                     // input
  Matrix<float> T2(T1.columns(), n1, /*columnMajor*/ true); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 blocks(ceildiv(n0, threads.x), ceildiv(n1, threads.y), 1);
  fillMatrices(T0, T1, T2, blocks, threads, deviceId);

  // Verify
  // shared memory: [t0 sub-matrix, t1 sub-matrix]
  threads.y = ceildiv(n1, threads.y * kTile);
  uint32_t smemSize = threads.x * threads.y * sizeof(float) * 2 * kTile;

  switch (strategy) {
  case Strategy::SynchronousCopy:
    bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
        n0, n1, n2, T0.data, T1.data, T2.data, kTile);
    break;
  case Strategy::AsynchronousCopy:
    bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
        n0, n1, n2, T0.data, T1.data, T2.data, kTile);
    break;
  default:
    throw std::runtime_error("Unknown strategy");
  };

  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    // Try different block sizes
    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = ceildiv(kMaxBlockDim, blockDim * kTile);
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y);
      smemSize = threads.x * threads.y * sizeof(float) * 2 * kTile;

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {

          switch (strategy) {
          case Strategy::SynchronousCopy:
            bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data, kTile);
            break;
          case Strategy::AsynchronousCopy:
            bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data, kTile);
            break;
          default:
            break;
          };

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  Strategy strategy = Strategy::AsynchronousCopy;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (argc > 2) {
    strategy = static_cast<Strategy>(atoi(argv[2]));
  }
  std::cout << "Using strategy " << static_cast<int>(strategy) << std::endl;

  run(deviceId, strategy);
  return 0;
}