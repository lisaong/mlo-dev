#include "hip/hip_runtime.h"
// Double-buffered banded matrix multiplication
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdint>
// #include <cuda/pipeline>
#include <hip/hip_runtime.h>

#define DEBUG 1
#include "constants.h"
#include "utils.h"

// https://developer.nvidia.com/blog/cooperative-groups/
namespace cg = cooperative_groups;

enum class Strategy { SynchronousCopy = 0, AsynchronousCopy = 1 };

__global__ void bandedMatMul_syncCopy(int n0, int n1, int n2, float *t0,
                                      const float *t1, const float *t2,
                                      int tileK) {

  int i, j, k;

  auto cta = cg::this_thread_block();

  extern __shared__ float t0_s[];                   // blockDim.x * blockDim.y
  float *t1_s = &t0_s[cta.size()];                  // blockDim.x * tileK
  float *t2_s = &t1_s[cta.dim_threads().x * tileK]; // blockDim.y * tileK

  // T1: 64xk, T2: kx16
  // 64 rows of T1, each thread copies k columns of T1
  // 16 columns of T2, each thread copies k rows of T2

  // for a blockDim.x x 1024 sub-matrix of T1 and a 1024 x blockDim.y submatrix
  // of T2
  //   copy a blockDim.x x k_tile of T1 into shared memory
  //   copy a k_tile x blockDim.y of T2 into shared memory (row-shifted by i)
  //   compute matmul and write to T0's shared memory

  const auto t1_rowStart = blockIdx.x * blockDim.x + threadIdx.x;
  const auto t1_rowStride = blockDim.x * gridDim.x;
  const auto t2_colStart = blockIdx.y * blockDim.y + threadIdx.y;
  const auto t2_colStride = blockDim.y * gridDim.y;

  for (i = t1_rowStart; i < n0; i += t1_rowStride) {
    for (j = t2_colStart; j < n1; j += t2_colStride) {
      auto smemIndex = threadIdx.x * blockDim.y + threadIdx.y;
      auto index = i * n1 + j;
      t0_s[smemIndex] = t0[index];
    }
  }
  cta.sync();

  for (i = t1_rowStart; i < n0; i += t1_rowStride) {
    // copy a blockDim.x x k_tile of T1 into shared memory
    for (k = 0; k < tileK; ++k) {
      auto smemIndex = threadIdx.x * tileK + k;
      auto index = i * n0 + blockIdx.y * tileK + k;
      t1_s[smemIndex] = t1[index];
    }

    for (j = t2_colStart; j < n1; j += t2_colStride) {
      // copy a k_tile x blockDim.y of T2 into shared memory (row-shifted by i)
      for (k = 0; k < tileK && (i + threadIdx.y * tileK + k) < n0; ++k) {
        auto smemIndex = threadIdx.y * tileK + k;
        auto index = (i + threadIdx.y * tileK + k) * n1 + j;

        t2_s[smemIndex] = t2[index];
      }
    }
  }
  cta.sync();

  // compute matmul and write to T0's shared memory
  for (i = t1_rowStart; i < n0; i += t1_rowStride) {
    for (j = t2_colStart; j < n1; j += t2_colStride) {
      for (k = 0; k < tileK; ++k) {
        t0_s[threadIdx.x * blockDim.y + threadIdx.y] +=
            t1_s[threadIdx.x * tileK + k] * t2_s[threadIdx.y * tileK + k];
      }

      cta.sync();

      // write to global memory
      t0[i * n1 + j] = t0_s[threadIdx.x * blockDim.y + threadIdx.y];
    }
  }
}

__global__ void bandedMatMul_asyncCopy(int n0, int n1, int n2, float *t0,
                                       const float *t1, const float *t2) {

  int i, j, k;

  auto cta = cg::this_thread_block();

  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size()];

  // each block will copy n0 / gridDim.x rows
  const auto numRows = n0 / gridDim.x;
  const auto rowOffset = blockIdx.x * numRows;
  for (int r = 0; r < numRows; ++r) {
    // copy a row of t0 and t1 into shared memory
    i = rowOffset + r;
    cg::memcpy_async(cta, t0_s, &t0[i * n1], sizeof(float) * cta.size());
    cg::memcpy_async(cta, t1_s, &t1[i * n1], sizeof(float) * cta.size());

    cg::wait(cta); // wait for copies to complete

    // compute the row, assumes the number of threads == row width
    j = threadIdx.x * blockDim.y + threadIdx.y;

    // treat t2 as column major
    for (k = 0; i + k < n1; ++k) {
      t0_s[j] += t1_s[j] * t2[(i + k) + j * n2];
    }
    cta.sync(); // wait for all threads to compute

    // write back to global memory
    t0[i * n1 + j] = t0_s[j];
    cta.sync(); // wait for all threads to consume
  }
}

void run(int deviceId, Strategy strategy) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);                                 // output
  BandedMatrix<float> T1(n0, kBandDim);                     // input
  Matrix<float> T2(T1.columns(), n1, /*columnMajor*/ true); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);
  fillMatrices(T0, T1, T2, blocks, threads, deviceId);

  // divide the inner dimension (k) among threads.y
  int tileK = n1 / threads.y;

  // hold tiles of T0, T1, and T2 in shared memory
  uint32_t smemSize = threads.x * threads.y * sizeof(float) +
                      threads.x * tileK * sizeof(float) +
                      threads.y * tileK * sizeof(float);

  // Verify
  switch (strategy) {
  case Strategy::SynchronousCopy:
    bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
        n0, n1, n2, T0.data, T1.data, T2.data, tileK);
    break;
  case Strategy::AsynchronousCopy:
    bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(n0, n1, n2, T0.data,
                                                          T1.data, T2.data);
    break;
  default:
    throw std::runtime_error("Unknown strategy");
  };

  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    // Try different block sizes
    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = kMaxBlockDim / blockDim;
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y);
      tileK = n1 / threads.y; // TODO: check
      smemSize = threads.x * threads.y * sizeof(float) +
                 threads.x * tileK * sizeof(float) +
                 threads.y * tileK * sizeof(float);

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {

          switch (strategy) {
          case Strategy::SynchronousCopy:
            bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data, tileK);
            break;
          case Strategy::AsynchronousCopy:
            bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data);
            break;
          default:
            break;
          };

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  Strategy strategy = Strategy::AsynchronousCopy;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (argc > 2) {
    strategy = static_cast<Strategy>(atoi(argv[2]));
  }
  std::cout << "Using strategy " << static_cast<int>(strategy) << std::endl;

  run(deviceId, strategy);
  return 0;
}