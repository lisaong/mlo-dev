#include "hip/hip_runtime.h"
// Double-buffered banded matrix multiplication
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdint>
#include <hip/hip_runtime.h>

// #define DEBUG 1
#include "constants.h"
#include "utils.h"

// https://developer.nvidia.com/blog/cooperative-groups/
namespace cg = cooperative_groups;

enum class Strategy { SynchronousCopy = 0, AsynchronousCopy = 1 };

__global__ void bandedMatMul_syncCopy(int n0, int n1, int n2, float *t0,
                                      const float *t1, const float *t2,
                                      int tile) {

  int i, j, k, jj;

  auto cta = cg::this_thread_block();

  // load the t0 and t1 sub-matrices into shared memory
  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size() * tile];

  const auto rowStart = blockIdx.x * blockDim.x + threadIdx.x;
  const auto rowStride = blockDim.x * gridDim.x;
  const auto colStart = blockIdx.y * blockDim.y + threadIdx.y;
  const auto colStride = blockDim.y * gridDim.y;

  for (i = rowStart; i < n0; i += rowStride) {
    for (j = colStart; j * tile < n1; j += colStride) {

      // for each thread, copy a column-tile of t0 and t1 into shared memory
      for (jj = 0; jj < tile; ++jj) {
        const auto smemIdx =
            threadIdx.x * blockDim.y * tile + threadIdx.y * tile + jj;
        t0_s[smemIdx] = t0[i * n1 + j * tile + jj];
        t1_s[smemIdx] = t1[i * n2 + j * tile + jj];
      }
    }
  }
  cta.sync();

  // compute
  for (i = rowStart; i < n0; i += rowStride) {
    for (j = colStart; j * tile < n1; j += colStride) {
      for (jj = 0; jj < tile; ++jj) {
        const auto smemIdx =
            threadIdx.x * blockDim.y * tile + threadIdx.y * tile + jj;

        // treat t2 as column major
        for (k = 0; i + k < n1; ++k) {
          t0_s[smemIdx] += t1_s[smemIdx] * t2[(i + k) + (j * tile + jj) * n2];
        }

        // write back to global memory
        t0[i * n1 + j * tile + jj] = t0_s[smemIdx];
      }
    }
  }
}

__global__ void bandedMatMul_asyncCopy(int n0, int n1, int n2, float *t0,
                                       const float *t1, const float *t2,
                                       int tile) {

  // extern __shared__ float t0_s[];

  // // cf. MatrixMulAsyncCopySingleStage in
  // //
  // https://github.com/NVIDIA/cuda-samples/blob/master/Samples/3_CUDA_Features/globalToShmemAsyncCopy/globalToShmemAsyncCopy.cu
  // auto cta = cg::this_thread_block();
  // float *t1_s = &t0_s[cta.size()];
  // int i, j, k;

  // // cooperatively copy each blockDim.x * blockDim.y tile of t0 and t1 to
  // shared
  // // memory
  // auto startX = blockIdx.x * blockDim.x + threadIdx.x;
  // auto strideX = blockDim.x * gridDim.x;
  // auto strideY = blockDim.y * gridDim.y;

  // for (i = startX; i < n0; i += strideX) {
  //   cg::memcpy_async(cta, t0_s, t0 + i * n1, sizeof(float) * strideY);
  //   cg::memcpy_async(cta, t1_s, t1 + i * n2, sizeof(float) * strideY);
  // }
  // cg::wait();

  // int columnOffset = cta.group_index().y * cta.dim_threads().y;
  // int columnStride = cta.dim_threads().y;
  // int smemOffset = cta.dim_threads().y;

  // for (i = startX; i < n0; i += strideX) {
  //   // copy a row
  //   int rowOffset = cta.group_index().x * cta.dim_threads().x + b;
  //   cg::memcpy_async(cta, t0_s + smemOffset * b,
  //                    &t0[rowOffset * n1 + columnOffset],
  //                    sizeof(float) * columnStride);
  //   cg::memcpy_async(cta, t1_s + smemOffset * b,
  //                    &t1[rowOffset * n2 + columnOffset],
  //                    sizeof(float) * columnStride);
  // }
  // cg::wait(cta);

  // // compute the row
  // i = cta.group_index().x * cta.dim_threads().x + cta.thread_index().x;
  // j = cta.group_index().y * cta.dim_threads().y + cta.thread_index().y;
  // for (k = 0; (i + k) < n1; ++k) {
  //   t0_s[cta.thread_index().x * cta.dim_threads().y + cta.thread_index().y]
  //   +=
  //       t1_s[cta.thread_index().x * cta.dim_threads().y +
  //            cta.thread_index().y] *
  //       t2[(i + k) + j * n2];
  // }

  // cg::sync(cta);

  // for (int b = 0; b < numRows; ++b) {
  //   // write back to t0 global memory
  //   int rowOffset = cta.group_index().x * cta.dim_threads().x + b;
  //   cg::memcpy_async(cta, &t0[rowOffset * n1 + columnOffset],
  //                    t0_s + smemOffset * b, sizeof(float) * columnStride);
  // }
}

void run(int deviceId, Strategy strategy) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);                                 // output
  BandedMatrix<float> T1(n0, kBandDim);                     // input
  Matrix<float> T2(T1.columns(), n1, /*columnMajor*/ true); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 blocks(ceildiv(n0, threads.x), ceildiv(n1, threads.y), 1);
  fillMatrices(T0, T1, T2, blocks, threads, deviceId);

  // Verify
  // shared memory: [t0 sub-matrix, t1 sub-matrix]
  uint32_t smemSize = threads.x * threads.y * sizeof(float) * 2;
  threads.y =
      ceildiv(threads.y, kTile); // each threadIdx.y will process kTile elements

  switch (strategy) {
  case Strategy::SynchronousCopy:
    bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
        n0, n1, n2, T0.data, T1.data, T2.data, kTile);
    break;
  case Strategy::AsynchronousCopy:
    bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
        n0, n1, n2, T0.data, T1.data, T2.data, kTile);
    break;
  default:
    throw std::runtime_error("Unknown strategy");
  };

  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    // Try different block sizes
    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = kMaxBlockDim / blockDim;
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y * kTile);
      smemSize = threads.x * threads.y * sizeof(float) * 2 * kTile;

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {

          switch (strategy) {
          case Strategy::SynchronousCopy:
            bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data, kTile);
            break;
          case Strategy::AsynchronousCopy:
            bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data, kTile);
            break;
          default:
            break;
          };

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  Strategy strategy = Strategy::AsynchronousCopy;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (argc > 2) {
    strategy = static_cast<Strategy>(atoi(argv[2]));
  }
  std::cout << "Using strategy " << static_cast<int>(strategy) << std::endl;

  run(deviceId, strategy);
  return 0;
}