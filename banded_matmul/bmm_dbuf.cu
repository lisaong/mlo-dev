#include "hip/hip_runtime.h"
// Double-buffered banded matrix multiplication
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdint>
#include <hip/hip_runtime.h>

#include "constants.h"
#include "utils.h"

// https://developer.nvidia.com/blog/cooperative-groups/
namespace cg = cooperative_groups;

enum class Strategy { SynchronousCopy = 0, AsynchronousCopy = 1 };

__global__ void bandedMatMul_syncCopy(int n0, int n1, int n2, float *t0,
                                      const float *t1, const float *t2) {

  int i, j, k;

  auto cta = cg::this_thread_block();

  // load the t0 and t1 sub-matrices into shared memory
  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size()];

  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.y * gridDim.y) {
      t0_s[threadIdx.x * blockDim.y + threadIdx.y] = t0[i * n1 + j];
      t1_s[threadIdx.x * blockDim.y + threadIdx.y] = t1[i * n2 + j];

      cta.sync();

      // treat t2 as column major
      for (k = 0; k < n2 && (i + k) < n0; ++k) {
        t0_s[threadIdx.x * blockDim.y + threadIdx.y] +=
            t1_s[threadIdx.x * blockDim.y + threadIdx.y] * t2[(i + k) + j * n2];
      }
      cta.sync();

      // write back to global memory
      t0[i * n1 + j] = t0_s[threadIdx.x * blockDim.y + threadIdx.y];
    }
  }
}

// __device__ void compute(int n0, int n1, int n2, int float *t0, const float
// *t1,
//                         const float *t2) {

//   for (int i = 0; i < n0; ++i) {
//     for (int j = 0; j < n1; ++j) {
//       for (int k = 0; (i + k) < n2; ++k) {
//         t0[i * n1 + j] += t1[i * n2 + k] * t2[(i + k) * n1 + j];
//       }
//     }
//   }
// }

__global__ void bandedMatMul_asyncCopy(int n0, int n1, int n2, float *t0,
                                       const float *t1, const float *t2) {

  auto cta = cg::this_thread_block();

  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size()];

  // load the t0 and t1 sub-matrices into shared memory
  cg::memcpy_async(cta, t0_s, &t0[cta.group_index().x * n1],
                   sizeof(float) * cta.size());
  cg::memcpy_async(cta, t1_s, &t1[cta.group_index().x * n2],
                   sizeof(float) * cta.size());
  cg::wait(cta);

  for (int i = cta.group_index().x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.y * gridDim.y) {

      // treat t2 as column major
      for (int k = 0; k < n2 && (i + k) < n0; ++k) {
        t0_s[threadIdx.x * blockDim.y + threadIdx.y] +=
            t1_s[threadIdx.x * blockDim.y + threadIdx.y] * t2[(i + k) + j * n2];
      }
    }
  }

  cta.sync();

  // write back to t0 global memory
  cg::memcpy_async(cta, &t0[cta.group_index().x * n1], t0_s,
                   sizeof(float) * cta.size());
  cg::wait(cta);
}

void run(int deviceId, Strategy strategy) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);             // output
  BandedMatrix<float> T1(n0, kBandDim); // input
  Matrix<float> T2(T1.columns(), n1);   // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  // dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 threads(16, 16, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);
  fillMatrices(T0, T1, T2, blocks, threads, deviceId);

  // Verify
  uint32_t smemSize;
  switch (strategy) {
  case Strategy::SynchronousCopy:
    // shared memory: [t0 sub-matrix, t1 sub-matrix]
    smemSize = threads.x * threads.y * sizeof(float) * 2;
    bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(n0, n1, n2, T0.data,
                                                         T1.data, T2.data);
    break;
  case Strategy::AsynchronousCopy:
    // shared memory: [t0 sub-matrix, t1 sub-matrix]
    smemSize = threads.x * threads.y * sizeof(float) * 2;
    bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(n0, n1, n2, T0.data,
                                                          T1.data, T2.data);
    break;
  default:
    throw std::runtime_error("Unknown strategy");
  };

  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    // Try different block sizes
    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = kMaxBlockDim / blockDim;
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y);

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {

          switch (strategy) {
          case Strategy::SynchronousCopy:
            smemSize = threads.x * threads.y * sizeof(float) * 2;
            bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data);
            break;
          case Strategy::AsynchronousCopy:
            smemSize = threads.x * threads.y * sizeof(float) * 2;
            bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data);
            break;
          default:
            break;
          };

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  Strategy strategy = Strategy::AsynchronousCopy;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (argc > 2) {
    strategy = static_cast<Strategy>(atoi(argv[2]));
  }
  std::cout << "Using strategy " << static_cast<int>(strategy) << std::endl;

  run(deviceId, strategy);
  return 0;
}