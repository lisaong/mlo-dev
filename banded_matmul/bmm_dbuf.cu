#include "hip/hip_runtime.h"
// Double-buffered banded matrix multiplication
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdint>
#include <hip/hip_runtime.h>

#define DEBUG 1
#include "constants.h"
#include "utils.h"

// https://developer.nvidia.com/blog/cooperative-groups/
namespace cg = cooperative_groups;

enum class Strategy { SynchronousCopy = 0, AsynchronousCopy = 1 };

__global__ void bandedMatMul_syncCopy(int n0, int n1, int n2, float *t0,
                                      const float *t1, const float *t2) {

  int i, j, k;

  auto cta = cg::this_thread_block();

  // load the t0 and t1 sub-matrices into shared memory
  extern __shared__ float t0_s[];
  float *t1_s = &t0_s[cta.size()];

  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < n0;
       i += blockDim.x * gridDim.x) {
    for (j = blockIdx.y * blockDim.y + threadIdx.y; j < n1;
         j += blockDim.y * gridDim.y) {
      t0_s[threadIdx.x * blockDim.y + threadIdx.y] = t0[i * n1 + j];
      t1_s[threadIdx.x * blockDim.y + threadIdx.y] = t1[i * n2 + j];

      cta.sync();

      // treat t2 as column major
      for (k = 0; k < n2 && (i + k) < n0; ++k) {
        t0_s[threadIdx.x * blockDim.y + threadIdx.y] +=
            t1_s[threadIdx.x * blockDim.y + threadIdx.y] * t2[(i + k) + j * n2];
      }
      cta.sync();

      // write back to global memory
      t0[i * n1 + j] = t0_s[threadIdx.x * blockDim.y + threadIdx.y];
    }
  }
}

__global__ void bandedMatMul_asyncCopy(int n0, int n1, int n2, float *t0,
                                       const float *t1, const float *t2) {

  extern __shared__ float t0_s[];

  // cf. MatrixMulAsyncCopySingleStage in
  // https://github.com/NVIDIA/cuda-samples/blob/master/Samples/3_CUDA_Features/globalToShmemAsyncCopy/globalToShmemAsyncCopy.cu
  auto cta = cg::this_thread_block();
  float *t1_s = &t0_s[cta.size()];

  // cooperatively copy each blockDim.x * blockDim.y tile of t0 and t1 to shared
  // memory
  int numRows = blockDim.x;
  int columnOffset = cta.group_index().y * blockDim.y;
  int columnStride = blockDim.y;

  for (int b = 0; b < numRows; ++b) {
    // copy the row
    int rowOffset = cta.group_index().x * blockDim.x + b;
    cg::memcpy_async(cta, t0_s, &t0[rowOffset * n1 + columnOffset],
                     sizeof(float) * columnStride);
    cg::memcpy_async(cta, t1_s, &t1[rowOffset * n2 + columnOffset],
                     sizeof(float) * columnStride);
    cg::wait(cta);
  }

  // compute the tile
  int i = cta.group_index().x * blockDim.x + threadIdx.x;
  int j = cta.group_index().y * blockDim.y + threadIdx.y;
  for (int k = 0; k < n2 && (i + k) < n0; ++k) {
    t0_s[threadIdx.x * blockDim.y + threadIdx.y] +=
        t1_s[threadIdx.x * blockDim.y + threadIdx.y] * t2[(i + k) + j * n2];
  }
  cta.sync();

  for (int b = 0; b < numRows; ++b) {
    // write back to t0 global memory
    int rowOffset = cta.group_index().x * blockDim.x + b;
    cg::memcpy_async(cta, &t0[rowOffset * n1 + columnOffset], t0_s,
                     sizeof(float) * columnStride);
    cg::wait(cta);
  }
}

void run(int deviceId, Strategy strategy) {

  const int n0 = N; // n0: number of rows in T0 and T1
  const int n1 = N; // n1: number of columns in T0 and T2
  const int n2 = N; // n2: inner or shared dimension, i.e.
                    //     number of columns in T1 and number of rows in T2

  Matrix<float> T0(n0, n1);                                 // output
  BandedMatrix<float> T1(n0, kBandDim);                     // input
  Matrix<float> T2(T1.columns(), n1, /*columnMajor*/ true); // input

  CHECK(hipMallocManaged(&T0.data, T0.size()));
  CHECK(hipMallocManaged(&T1.data, T1.size()));
  CHECK(hipMallocManaged(&T2.data, T2.size()));

  // Initialize
  dim3 threads(kBlockDimX, kMaxBlockDim / kBlockDimX, 1);
  dim3 blocks(n0 / threads.x, n1 / threads.y, 1);
  fillMatrices(T0, T1, T2, blocks, threads, deviceId);

  // Verify
  uint32_t smemSize;
  switch (strategy) {
  case Strategy::SynchronousCopy:
    // shared memory: [t0 sub-matrix, t1 sub-matrix]
    smemSize = threads.x * threads.y * sizeof(float) * 2;
    bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(n0, n1, n2, T0.data,
                                                         T1.data, T2.data);
    break;
  case Strategy::AsynchronousCopy:
    // shared memory: [t0 sub-matrix, t1 sub-matrix]
    smemSize = threads.x * threads.y * sizeof(float) * 2;
    bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(n0, n1, n2, T0.data,
                                                          T1.data, T2.data);
    break;
  default:
    throw std::runtime_error("Unknown strategy");
  };

  CHECK(hipGetLastError());
  CHECK(hipDeviceSynchronize());

  if (checkCorrectness(n0, n1, n2, T0, T1, T2)) {

    // Benchmark
    hipEvent_t _start;
    hipEvent_t _stop;
    hipEventCreate(&_start);
    hipEventCreate(&_stop);

    // Try different block sizes
    std::cout << "GridDim,BlockDim,FLOPS,GFLOPS" << std::endl;

    for (uint32_t blockDim = kBlockDimX; blockDim <= kBlockDimXMax;
         blockDim += kBlockDimXStep) {

      threads.x = blockDim;
      threads.y = kMaxBlockDim / blockDim;
      blocks.x = ceildiv(n0, threads.x);
      blocks.y = ceildiv(n1, threads.y);

      try {
        double elapsedTimeMilliseconds = 0.0f;
        uint64_t iterations = 0;
        float duration = 0.0f;

        // Runs the function until 10 seconds has elapsed
        hipEventRecord(_start);
        while (elapsedTimeMilliseconds < kTimelimit) {

          switch (strategy) {
          case Strategy::SynchronousCopy:
            smemSize = threads.x * threads.y * sizeof(float) * 2;
            bandedMatMul_syncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data);
            break;
          case Strategy::AsynchronousCopy:
            smemSize = threads.x * threads.y * sizeof(float) * 2;
            bandedMatMul_asyncCopy<<<blocks, threads, smemSize>>>(
                n0, n1, n2, T0.data, T1.data, T2.data);
            break;
          default:
            break;
          };

          CHECK(hipGetLastError());
          CHECK(hipDeviceSynchronize());

          hipEventRecord(_stop);
          hipEventSynchronize(_stop);
          hipEventElapsedTime(&duration, _start, _stop);
          elapsedTimeMilliseconds += duration;
          iterations++;
        }

        const double flops = iterations * kNumberOfOps /
                             (elapsedTimeMilliseconds / kMillisecondsInSeconds);
        std::cout << blocks.x << "," << threads.x << "," << flops << ","
                  << flops / 1e9 << std::endl;
      } catch (const std::exception &e) {
        std::cout << "Skipping Blocksize: " << blockDim << ", " << e.what()
                  << std::endl;
        continue;
      }
    }

    hipEventDestroy(_start);
    hipEventDestroy(_stop);
  }

  hipFree(T0.data);
  hipFree(T1.data);
  hipFree(T2.data);
}

int main(int argc, const char **argv) {
  int deviceId;
  Strategy strategy = Strategy::AsynchronousCopy;

  if (argc > 1) {
    deviceId = atoi(argv[1]);
    CHECK(hipSetDevice(deviceId));
  } else {
    CHECK(hipGetDevice(&deviceId));
  }
  std::cout << "Using device " << deviceId << std::endl;

  if (argc > 2) {
    strategy = static_cast<Strategy>(atoi(argv[2]));
  }
  std::cout << "Using strategy " << static_cast<int>(strategy) << std::endl;

  run(deviceId, strategy);
  return 0;
}