#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>

#include "inc/timed_region.h"

using float16_t = _Float16;

#ifndef HIP_ASSERT
#define HIP_ASSERT(x) (assert((x) == hipSuccess))
#endif

#define CDIV(n, block) (n + block - 1) / block

enum class Strategy
{
    Naive = 0,
    Tiled = 1
};

template <typename T>
__global__ void init(T *a, uint64_t M, uint64_t N)
{
    const int i = blockDim.y * blockIdx.y + threadIdx.y;
    const int j = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M && j < N)
    {
        a[i * N + j] = static_cast<T>(i * N + j) / static_cast<T>(N * M / 4);
    }
}

// cf. https://gitlab.com/syifan/hipbookexample/-/blob/main/Chapter5/MatrixMultiplication/main.cpp

__global__ void matrixMultiplyTiled(float16_t *A, float16_t *B, float *C, uint64_t M, uint64_t N, uint64_t K, int tileSize)
{
    // C[i, j] += A[i, k] * B[k, j]
    // (M, N)    (M, K)    (K, N)
    //   where y => rows (i), x => colummns (j)

    extern __shared__ float subTileM[];
    float *subTileN = &subTileM[tileSize * tileSize];
}

__global__ void matrixMultiplyNaive(float16_t *A, float16_t *B, float *C, uint64_t M, uint64_t N, uint64_t K)
{
    // C[i, j] += A[i, k] * B[k, j]
    // (M, N)    (M, K)    (K, N)
    //   where y => rows (i), x => colummns (j)
    const int i = blockDim.y * blockIdx.y + threadIdx.y;
    const int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < M && j < N)
    {
        // multiply then sum along the k dimension
        float sum = 0.0f;
        for (int k = 0; k < K; ++k)
        {
            sum += A[i * K + k] * B[k * N + j];
        }
        C[i * N + j] = sum;
    }
}

#ifdef VERIFY
void matrixMultiplyCPU(float16_t *A, float16_t *B, float *C, uint64_t M, uint64_t N, uint64_t K)
{
    // C[i, j] += A[i, k] * B[k, j]
    // (M, N)    (M, K)    (K, N)
    //   where y => rows (i), x => colummns (j)
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k)
            {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}
#endif // VERIFY

int run(int deviceId, int tileSize, Strategy strategy)
{
#ifdef VERIFY
    constexpr uint64_t M = 64;
#else
    constexpr uint64_t M = 2 << 16;
#endif // VERIFY
    constexpr uint64_t N = M;
    constexpr uint64_t K = M;

    const dim3 numThreads(tileSize, tileSize, 1);
    const dim3 numBlocks(CDIV(M, numThreads.x), CDIV(N, numThreads.y), 1);

    // alloc
    float16_t *d_a;
    float16_t *d_b;
    float *d_c;
    HIP_ASSERT(hipMallocManaged(&d_a, M * K * sizeof(float16_t)));
    HIP_ASSERT(hipMallocManaged(&d_b, K * N * sizeof(float16_t)));
    HIP_ASSERT(hipMallocManaged(&d_c, M * N * sizeof(float)));

    HIP_ASSERT(hipMemPrefetchAsync(d_a, M * K * sizeof(float16_t), deviceId));
    HIP_ASSERT(hipMemPrefetchAsync(d_b, K * N * sizeof(float16_t), deviceId));
    HIP_ASSERT(hipMemPrefetchAsync(d_c, M * N * sizeof(float), deviceId));

    init<<<numBlocks, numThreads>>>(d_a, M, K);
    init<<<numBlocks, numThreads>>>(d_b, K, N);
    hipDeviceSynchronize();

    std::stringstream ss;
    ss << numBlocks.x << "," << numThreads.x; // BUGBUG: assumes square sizes
    if (strategy == Strategy::Naive)
    {
        TimedRegion r(ss.str());

        matrixMultiplyNaive<<<numBlocks, numThreads>>>(d_a, d_b, d_c, M, N, K);
        hipDeviceSynchronize();
    }
    else
    {
        int sharedMemorySize = tileSize * tileSize * sizeof(float) * 2;

        TimedRegion r(ss.str());

        matrixMultiplyTiled<<<numBlocks, numThreads, sharedMemorySize>>>(d_a, d_b, d_c, M, N, K, tileSize);
        hipDeviceSynchronize();
    }

#ifdef VERIFY
    {
        float *cVerify = new float[M * N];
        matrixMultiplyCPU(d_a, d_b, cVerify, M, N, K);

        for (int i = 0; i < M; ++i)
        {
            for (int j = 0; j < N; ++j)
            {
                // std::cout << d_c[i * N + j] << ", expected "
                //           << cVerify[i * N + j] << std::endl;

                if (abs(d_c[i * N + j] - cVerify[i * N + j]) > 1e-5)
                {
                    std::cout << "Error: C[" << i << ", " << j << "] = "
                              << d_c[i * N + j] << ", expected "
                              << cVerify[i * N + j] << std::endl;
                    return -1;
                }
            }
        }

        delete[] cVerify;
    }
#endif // VERIFY

    HIP_ASSERT(hipFree(d_a));
    HIP_ASSERT(hipFree(d_b));
    HIP_ASSERT(hipFree(d_c));

    return 0;
}

int main(int argc, const char **argv)
{
    int deviceId = 0;
    HIP_ASSERT(hipGetDevice(&deviceId));

    int supportsManagedMemory = 0;
    HIP_ASSERT(hipDeviceGetAttribute(&supportsManagedMemory,
                                     hipDeviceAttributeManagedMemory, deviceId));

    if (supportsManagedMemory == 0)
    {
        std::cout << "Managed memory is not supported for device " << deviceId << std::endl;
        return -1;
    }

    Strategy strategy = Strategy::Tiled;
    if (argc > 1)
    {
        strategy = static_cast<Strategy>(atoi(argv[1]));
    }

    std::cout << "grid_size,block_size,elapsed_msec" << std::endl;
    int result = 0;
    for (int numThreads = 32; numThreads <= 2500 && result == 0; numThreads += 32)
    {
        result = run(deviceId, numThreads, strategy);
    }
    return result;
}